#include "add.cu"

int main() {
	instance_t          *instances, *gpuInstances;
	cgbn_error_report_t *report;

	printf("Genereating instances ...\n");
	instances=generate_instances(INSTANCES);

	printf("Copying instances to the GPU ...\n");
	CUDA_CHECK(hipSetDevice(0));
	CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*INSTANCES));
	CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*INSTANCES, hipMemcpyHostToDevice));

	// create a cgbn_error_report for CGBN to report back errors
	CUDA_CHECK(cgbn_error_report_alloc(&report));

	printf("Running GPU kernel ...\n");
	// launch with 32 threads per instance, 128 threads (4 instances) per block
	kernel_add<<<(INSTANCES+3)/4, 128>>>(report, gpuInstances, INSTANCES);

	// error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
	CUDA_CHECK(hipDeviceSynchronize());
	CGBN_CHECK(report);

	// copy the instances back from gpuMemory
	printf("Copying results back to CPU ...\n");
	CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*INSTANCES, hipMemcpyDeviceToHost));

	printf("Verifying the results ...\n");
	verify_results(instances, INSTANCES);

	// clean up
	free(instances);
	CUDA_CHECK(hipFree(gpuInstances));
	CUDA_CHECK(cgbn_error_report_free(report));
}